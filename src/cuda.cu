/**
 * Matthew Smith
 * github.com/mattonaise
 * Created on Oct 05, 2019
 *
 * cuda.cu
 **/

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "cuda.hpp"

// defining kernel in seperate source file for clarity
#include "kernel.cu"

/* 
 * CUDA error-checking function and macro - from CUDA BY EXAMPLE
 */
static void HandleError(hipError_t err, const char* file, int line)
{
    if(err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err) (HandleError( err, __FILE__, __LINE__ ))

void cudaInit()
{
    printf("===CUDA INITIALIZATION===\n");

    // Select CUDA device with compute capability >=3.0
    int device;
    hipDeviceProp_t prop;
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 3;
    prop.minor = 0;
    HANDLE_ERROR( hipChooseDevice(&device, &prop) );

    cudaPrintDeviceProperties(device);
}

void cudaPrintDeviceProperties(int device)
{
    // Query device properties
    hipDeviceProp_t prop;
    int driverVersion, runtimeVersion;
    HANDLE_ERROR( hipGetDeviceProperties(&prop, device) );
    HANDLE_ERROR( hipDriverGetVersion(&driverVersion) );
    HANDLE_ERROR( hipRuntimeGetVersion(&runtimeVersion) );

    // Print device properties
    printf("\tDevice Name: %s\n", prop.name);
    printf("\tCUDA Driver Version / Runtime Version: %d.%d / %d.%d\n",
            driverVersion / 1000, (driverVersion % 100) / 10,
            runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("\tCompute Capability: %d.%d\n", prop.major, prop.minor);
    printf("\tTotal Global Memory: %ld bytes\n", prop.totalGlobalMem);
    printf("\tNumber of Multiprocessors: %d\n", prop.multiProcessorCount);
    printf("\tMaximum Threads per Multiprocessor: %d\n",
            prop.maxThreadsPerMultiProcessor);
    printf("\tTotal Number of Threads: %d\n", prop.multiProcessorCount *
            prop.maxThreadsPerMultiProcessor);
    printf("\tMaximum Threads per Block: %d\n", prop.maxThreadsPerBlock);
}

void* cudaRegisterBuffer(GLuint buf)
{
    printf("CUDA: Registering OpenGL buffer %d\n", buf);
    hipGraphicsResource* resource = nullptr;
    HANDLE_ERROR( hipGraphicsGLRegisterBuffer(&resource, buf, 
            cudaGraphicsMapFlagsNone) );
    return resource;
}

void cudaUnregisterResource(void* resource)
{
    printf("CUDA: Unregistering resource %p\n", resource);
    HANDLE_ERROR( hipGraphicsUnregisterResource((hipGraphicsResource*)
            resource) );
}

void cudaMapResource(void* resource, void** devicePtr, size_t* size)
{
    // map CUDA resource and get device pointer and size
    HANDLE_ERROR( hipGraphicsMapResources(1, (hipGraphicsResource**)
            &resource) );
    void* devicePtr_ = nullptr;
    size_t size_ = 0;
    HANDLE_ERROR( hipGraphicsResourceGetMappedPointer(&devicePtr_, &size_,
            (hipGraphicsResource*) resource) );
    printf("CUDA: Mapped resource returned pointer %p with size %d\n",
            devicePtr_, size_);

    // ensure devicePtr_ and size_ are valid
    assert(devicePtr_ != nullptr);
    assert(size_ != 0);
    
    // set pointer values
    *devicePtr = devicePtr_;
    *size = size_;
}

void cudaUnmapResource(void* resource)
{
    HANDLE_ERROR( hipGraphicsUnmapResources(1, (hipGraphicsResource**)
            &resource) );
}

void cudaRunKernel(void* devicePtr, size_t size)
{
    kernel<<<5,5>>>();

    // handle any synchronous and asynchronous kernel errors
    HANDLE_ERROR( hipGetLastError() );
    HANDLE_ERROR( hipDeviceSynchronize() );
}

void cudaShutdown()
{
    printf("===CUDA SHUTDOWN===\n");
}
