/**
 * Matthew Smith
 * github.com/mattonaise
 * Created on Oct 05, 2019
 *
 * cuda.cu
 **/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "cuda.hpp"

/* 
 * CUDA error-checking function and macro - from CUDA BY EXAMPLE
 */
static void HandleError(hipError_t err, const char* file, int line)
{
    if(err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err) (HandleError( err, __FILE__, __LINE__ ))

void cudaInit()
{
    printf("===CUDA INITIALIZATION===\n");

    // Select CUDA device with compute capability >=3.0
    int device;
    hipDeviceProp_t prop;
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 3;
    prop.minor = 0;
    HANDLE_ERROR( hipChooseDevice(&device, &prop) );

    cudaPrintDeviceProperties(device);
}

void cudaPrintDeviceProperties(int device)
{
    // Query device properties
    hipDeviceProp_t prop;
    int driverVersion, runtimeVersion;
    HANDLE_ERROR( hipGetDeviceProperties(&prop, device) );
    HANDLE_ERROR( hipDriverGetVersion(&driverVersion) );
    HANDLE_ERROR( hipRuntimeGetVersion(&runtimeVersion) );

    // Print device properties
    printf("\tDevice Name: %s\n", prop.name);
    printf("\tCUDA Driver Version / Runtime Version: %d.%d / %d.%d\n",
            driverVersion / 1000, (driverVersion % 100) / 10,
            runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("\tCompute Capability: %d.%d\n", prop.major, prop.minor);
    printf("\tTotal Global Memory: %ld bytes\n", prop.totalGlobalMem);
    printf("\tNumber of Multiprocessors: %d\n", prop.multiProcessorCount);
    printf("\tMaximum Threads per Multiprocessor: %d\n",
            prop.maxThreadsPerMultiProcessor);
    printf("\tTotal Number of Threads: %d\n", prop.multiProcessorCount *
            prop.maxThreadsPerMultiProcessor);
    printf("\tMaximum Threads per Block: %d\n", prop.maxThreadsPerBlock);
}

void* cudaRegisterBuffer(GLuint buf)
{
    printf("CUDA: Registering OpenGL buffer %d\n", buf);
    hipGraphicsResource* resource = nullptr;
    HANDLE_ERROR( hipGraphicsGLRegisterBuffer(&resource, buf, 
            cudaGraphicsMapFlagsNone) );
    return resource;
}

void cudaUnregisterResource(void* resource)
{
    printf("CUDA: Unregistering resource %p\n", resource);
    HANDLE_ERROR( hipGraphicsUnregisterResource((hipGraphicsResource*)
            resource) );
}

void* cudaMapResource(void* resource)
{
    HANDLE_ERROR( hipGraphicsMapResources(1, (hipGraphicsResource**)
            &resource) );
    void* devicePtr = nullptr;
    size_t size;
    HANDLE_ERROR( hipGraphicsResourceGetMappedPointer(&devicePtr, &size,
            (hipGraphicsResource*) resource) );
    return devicePtr;
}

void cudaUnmapResource(void* resource)
{
    HANDLE_ERROR( hipGraphicsUnmapResources(1, (hipGraphicsResource**)
            &resource) );
}

void cudaShutdown()
{
    printf("===CUDA SHUTDOWN===\n");
}
