#include "hip/hip_runtime.h"
/**
 * Matthew Smith
 * github.com/mattonaise
 * Created on Oct 05, 2019
 *
 * cuda.cu
 **/

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand/hiprand_kernel.h>

#include "cuda.hpp"
#include "mapping.hpp"

// defining kernel in seperate source file for clarity
#include "kernel.cu"

/* 
 * CUDA error-checking function and macro - from CUDA BY EXAMPLE
 */
static void HandleError(hipError_t err, const char* file, int line,
    bool abort = true)
{
    if(err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        if(abort)
            exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err) (HandleError( err, __FILE__, __LINE__ ))

void cudaInit()
{
    printf("===CUDA INITIALIZATION===\n");

    // Select CUDA device with compute capability >=6.0
    // Compute capability 6.0 needed for unified memory
    int device;
    hipDeviceProp_t prop;
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 6;
    prop.minor = 0;
    HANDLE_ERROR( hipChooseDevice(&device, &prop) );

    cudaPrintDeviceProperties(device);
}

void cudaPrintDeviceProperties(int device)
{
    // Query device properties
    hipDeviceProp_t prop;
    int driverVersion, runtimeVersion;
    HANDLE_ERROR( hipGetDeviceProperties(&prop, device) );
    HANDLE_ERROR( hipDriverGetVersion(&driverVersion) );
    HANDLE_ERROR( hipRuntimeGetVersion(&runtimeVersion) );

    // Print device properties
    printf("\tDevice Name: %s\n", prop.name);
    printf("\tCUDA Driver Version / Runtime Version: %d.%d / %d.%d\n",
            driverVersion / 1000, (driverVersion % 100) / 10,
            runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("\tCompute Capability: %d.%d\n", prop.major, prop.minor);
    printf("\tTotal Global Memory: %ld bytes\n", prop.totalGlobalMem);
    printf("\tNumber of Multiprocessors: %d\n", prop.multiProcessorCount);
    printf("\tMaximum Threads per Multiprocessor: %d\n",
            prop.maxThreadsPerMultiProcessor);
    printf("\tTotal Number of Threads: %d\n", prop.multiProcessorCount *
            prop.maxThreadsPerMultiProcessor);
    printf("\tMaximum Threads per Block: %d\n", prop.maxThreadsPerBlock);
}

void* cudaRegisterBuffer(GLuint buf)
{
    hipGraphicsResource* resource = nullptr;
    HANDLE_ERROR( hipGraphicsGLRegisterBuffer(&resource, buf, 
            cudaGraphicsMapFlagsNone) );
    return resource;
}

void cudaUnregisterResource(void* resource)
{
    HANDLE_ERROR( hipGraphicsUnregisterResource((hipGraphicsResource*)
            resource) );
}

void cudaMapResource(void* resource, void** devicePtr, size_t* size)
{
    // map CUDA resource and get device pointer and size
    HANDLE_ERROR( hipGraphicsMapResources(1, (hipGraphicsResource**)
            &resource) );
    void* devicePtr_ = nullptr;
    size_t size_ = 0;
    HANDLE_ERROR( hipGraphicsResourceGetMappedPointer(&devicePtr_, &size_,
            (hipGraphicsResource*) resource) );

    // ensure devicePtr_ and size_ are valid
    assert(devicePtr_ != nullptr);
    assert(size_ != 0);
    
    // set pointer values
    *devicePtr = devicePtr_;
    *size = size_;
}

void cudaUnmapResource(void* resource)
{
    HANDLE_ERROR( hipGraphicsUnmapResources(1, (hipGraphicsResource**)
            &resource) );
}

void cudaAllocateMapping(mapping** mapping, int numMappings)
{
    HANDLE_ERROR( hipMallocManaged(mapping, numMappings * sizeof(mapping)) );
}

void cudaDeallocateMapping(mapping* mapping)
{
    HANDLE_ERROR( hipFree(mapping) );
}

void cudaRunKernel(void* d_pointData, int numPoints,
        mapping* d_mappings, int numMappings)
{
    // calculate block numbers and block size
    int blockSize = 256;
    //int iterations = 256;
    //int numBlocks = (numPoints + blockSize - 1) / (blockSize * iterations); 
    int numBlocks = 1;
    printf("CUDA: Running kernel (%d block(s), %d threads per block) - ",
            numBlocks, blockSize);
    
    // set up CUDA events for timing the kernel
    hipEvent_t start, stop;
    HANDLE_ERROR( hipEventCreate(&start) );
    HANDLE_ERROR( hipEventCreate(&stop) );
    
    HANDLE_ERROR( hipEventRecord(start) );
    kernel<<<numBlocks, blockSize, numMappings * sizeof(mapping)>>>
        ((float4*)d_pointData, numPoints, d_mappings, numMappings);
    HANDLE_ERROR( hipEventRecord(stop) );
 
    // handle any synchronous and asynchronous kernel errors
    HANDLE_ERROR( hipGetLastError() );
    HANDLE_ERROR( hipDeviceSynchronize() );

    // record and print kernel timing
    HANDLE_ERROR( hipEventSynchronize(stop) );
    float milliseconds = 0;
    HANDLE_ERROR( hipEventElapsedTime(&milliseconds, start, stop) );
    printf("%f ms\n", milliseconds);
}

void cudaShutdown()
{
    printf("===CUDA SHUTDOWN===\n");
}
