#include "hip/hip_runtime.h"
/**
 * Matthew Smith
 * github.com/mattonaise
 * Created on Oct 05, 2019
 *
 * cuda.cu
 **/

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "cuda.hpp"

// defining kernel in seperate source file for clarity
#include "kernel.cu"

/* 
 * CUDA error-checking function and macro - from CUDA BY EXAMPLE
 */
static void HandleError(hipError_t err, const char* file, int line)
{
    if(err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err) (HandleError( err, __FILE__, __LINE__ ))

void cudaInit()
{
    printf("===CUDA INITIALIZATION===\n");

    // Select CUDA device with compute capability >=3.0
    int device;
    hipDeviceProp_t prop;
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 3;
    prop.minor = 0;
    HANDLE_ERROR( hipChooseDevice(&device, &prop) );

    cudaPrintDeviceProperties(device);
}

void cudaPrintDeviceProperties(int device)
{
    // Query device properties
    hipDeviceProp_t prop;
    int driverVersion, runtimeVersion;
    HANDLE_ERROR( hipGetDeviceProperties(&prop, device) );
    HANDLE_ERROR( hipDriverGetVersion(&driverVersion) );
    HANDLE_ERROR( hipRuntimeGetVersion(&runtimeVersion) );

    // Print device properties
    printf("\tDevice Name: %s\n", prop.name);
    printf("\tCUDA Driver Version / Runtime Version: %d.%d / %d.%d\n",
            driverVersion / 1000, (driverVersion % 100) / 10,
            runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("\tCompute Capability: %d.%d\n", prop.major, prop.minor);
    printf("\tTotal Global Memory: %ld bytes\n", prop.totalGlobalMem);
    printf("\tNumber of Multiprocessors: %d\n", prop.multiProcessorCount);
    printf("\tMaximum Threads per Multiprocessor: %d\n",
            prop.maxThreadsPerMultiProcessor);
    printf("\tTotal Number of Threads: %d\n", prop.multiProcessorCount *
            prop.maxThreadsPerMultiProcessor);
    printf("\tMaximum Threads per Block: %d\n", prop.maxThreadsPerBlock);
}

void* cudaRegisterBuffer(GLuint buf)
{
    hipGraphicsResource* resource = nullptr;
    HANDLE_ERROR( hipGraphicsGLRegisterBuffer(&resource, buf, 
            cudaGraphicsMapFlagsNone) );
    return resource;
}

void cudaUnregisterResource(void* resource)
{
    HANDLE_ERROR( hipGraphicsUnregisterResource((hipGraphicsResource*)
            resource) );
}

void cudaMapResource(void* resource, void** devicePtr, size_t* size)
{
    // map CUDA resource and get device pointer and size
    HANDLE_ERROR( hipGraphicsMapResources(1, (hipGraphicsResource**)
            &resource) );
    void* devicePtr_ = nullptr;
    size_t size_ = 0;
    HANDLE_ERROR( hipGraphicsResourceGetMappedPointer(&devicePtr_, &size_,
            (hipGraphicsResource*) resource) );

    // ensure devicePtr_ and size_ are valid
    assert(devicePtr_ != nullptr);
    assert(size_ != 0);
    
    // set pointer values
    *devicePtr = devicePtr_;
    *size = size_;
}

void cudaUnmapResource(void* resource)
{
    HANDLE_ERROR( hipGraphicsUnmapResources(1, (hipGraphicsResource**)
            &resource) );
}

void cudaRunKernel(void* devicePtr, size_t size)
{
    int blocks = 1;
    size_t newSize = size / sizeof(float3);
    printf("CUDA: Running kernel (%d blocks, %d threads per block)\n",
            blocks, newSize);
    kernel<<<blocks,newSize>>>((float3*)devicePtr, newSize);

    // handle any synchronous and asynchronous kernel errors
    HANDLE_ERROR( hipGetLastError() );
    HANDLE_ERROR( hipDeviceSynchronize() );
}

void cudaShutdown()
{
    printf("===CUDA SHUTDOWN===\n");
}
