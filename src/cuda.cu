/**
 * Matthew Smith
 * github.com/mattonaise
 * Created on Oct 05, 2019
 *
 * cuda.cu
 **/

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "cuda.hpp"

/* CUDA error-checking function and macro - from CUDA BY EXAMPLE */
static void HandleError(hipError_t err, const char* file, int line)
{
    if(err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err) (HandleError( err, __FILE__, __LINE__ ))

void cudaInit()
{
    printf("===CUDA INITIALIZATION===\n");

    /* Query device count */
    int nDevices;
    HANDLE_ERROR( hipGetDeviceCount(&nDevices) );
    printf("Total devices Found: %d\n", nDevices);

    /* Print device info */
    for(int i = 0; i < nDevices; i++)
    {
        /* Query device properties */
        hipDeviceProp_t prop;
        int driverVersion, runtimeVersion;
        HANDLE_ERROR( hipGetDeviceProperties(&prop, i) );
        HANDLE_ERROR( hipSetDevice(i) );
        HANDLE_ERROR( hipDriverGetVersion(&driverVersion) );
        HANDLE_ERROR( hipRuntimeGetVersion(&runtimeVersion) );

        /* Print device properties */
        printf("Device Number: %d\n", i);
        printf("\tDevice Name: %s\n", prop.name);
        printf("\tCUDA Driver Version / Runtime Version: %d.%d / %d.%d\n",
                driverVersion / 1000, (driverVersion % 100) / 10,
                runtimeVersion / 1000, (runtimeVersion % 100) / 10);
        printf("\tCompute Capability: %d.%d\n", prop.major, prop.minor);
        printf("\tTotal Global Memory: %ld bytes\n", prop.totalGlobalMem);
        printf("\tNumber of Multiprocessors: %d\n", prop.multiProcessorCount);
        printf("\tMaximum Threads per Multiprocessor: %d\n",
                prop.maxThreadsPerMultiProcessor);
        printf("\tTotal Number of Threads: %d\n", prop.multiProcessorCount *
                prop.maxThreadsPerMultiProcessor);
        printf("\tMaximum Threads per Block: %d\n", prop.maxThreadsPerBlock);
    }
}

void cudaShutdown()
{
    printf("===CUDA SHUTDOWN===\n");
}
