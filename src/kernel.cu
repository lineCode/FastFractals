#include "hip/hip_runtime.h"
/**
 * Matthew Smith
 * github.com/mattonaise
 * Created on Oct 11, 2019
 *
 * kernel.cu
 **/

__global__ void kernel(float4* ptr, int numPoints)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // If needed for performance, move hiprand_init to seperate kernel and store
    // states in device memory
    hiprandState state;
    hiprand_init((unsigned long long) clock(), index, 0, &state);

    for(int i = index; i < numPoints; i += stride)
    {
        ptr[i].x = hiprand_uniform(&state) * 2 - 1;
        ptr[i].y = hiprand_uniform(&state) * 2 - 1;
        ptr[i].z = hiprand_uniform(&state) * 2 - 1;
        ptr[i].w = 1.0f;
    }
}
